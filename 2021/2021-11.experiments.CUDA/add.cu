// Based on "An Even Easier Introduction to CUDA", by Mark Harris
// https://developer.nvidia.com/blog/even-easier-introduction-cuda/


#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// CUDA Kernel function to add the elements of two arrays on the GPU
__global__
void add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++)
      y[i] = x[i] + y[i];
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main(int argc, char *argv[])
{
  int p = 6;
  if (argc == 2)
  {
      p = atoi(argv[1]);
  }
  long N = pow(10, p);
  long size = N*sizeof(float);
  std::cout << "N: " << N << std::endl;

  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  gpuErrchk(hipMallocManaged(&x, size));
  gpuErrchk(hipMallocManaged(&y, size));

  // initialize x and y arrays on the host
  for (long i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on N elements on the GPU
  add<<<1, 1>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  gpuErrchk(hipDeviceSynchronize());

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (long i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  gpuErrchk(hipFree(x));
  gpuErrchk(hipFree(y));
  
  return 0;
}