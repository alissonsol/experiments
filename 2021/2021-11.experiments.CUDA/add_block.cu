#include "hip/hip_runtime.h"
// Based on "An Even Easier Introduction to CUDA", by Mark Harris
// https://developer.nvidia.com/blog/even-easier-introduction-cuda/

#include <iostream>
#include <math.h>

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int index = threadIdx.x;
  int stride = blockDim.x;
  for (int i = index; i < n; i += stride)
      y[i] = x[i] + y[i];
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main(int argc, char *argv[])
{
  int p = 6;
  if (argc == 2)
  {
      p = atoi(argv[1]);
  }
  long N = pow(10, p);
  long size = N*sizeof(float);
  std::cout << "N: " << N << std::endl;

  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  gpuErrchk(hipMallocManaged(&x, size));
  gpuErrchk(hipMallocManaged(&y, size));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on N elements on the GPU
  add<<<1, 256>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  gpuAssert(hipDeviceSynchronize());

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  gpuAssert(hipFree(x));
  gpuAssert(hipFree(y));
  
  return 0;
}