// Based on "An Even Easier Introduction to CUDA", by Mark Harris
// https://developer.nvidia.com/blog/even-easier-introduction-cuda/


#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main(int argc, char *argv[])
{
  int p = 6;
  if (argc == 2)
  {
      p = atoi(argv[1]);
  }
  long N = pow(10, p);
  long size = N*sizeof(float);
  std::cout << "N: " << N << std::endl;

  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  gpuErrchk(hipMallocManaged(&x, size));
  gpuErrchk(hipMallocManaged(&y, size));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on N elements on the GPU
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  add<<<numBlocks, blockSize>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  gpuErrchk(hipDeviceSynchronize());

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  gpuErrchk(hipFree(x));
  gpuErrchk(hipFree(y));
  
  return 0;
}